
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "hipblas.h"

extern "C" int f_cublasCreate(hipblasHandle_t **handle)
{
    *handle = (hipblasHandle_t*)malloc(sizeof(hipblasHandle_t));
    return hipblasCreate(*handle);
}

extern "C" int f_cublasZgemm(hipblasHandle_t *handle,
               hipblasOperation_t transa, hipblasOperation_t transb, 
              int m, int n, int k, 
              const hipDoubleComplex *alpha,
              const hipDoubleComplex *A, int lda, 
              const hipDoubleComplex *B, int ldb,
              const hipDoubleComplex *beta, 
              hipDoubleComplex *C, int ldc)
{
    return hipblasZgemm(*handle,transa,transb,m,n,k,alpha,A,lda,B,ldb,beta,C,ldc);
}

extern "C" int f_cublasZgemmBatched(hipblasHandle_t *handle,
               hipblasOperation_t transa, hipblasOperation_t transb,
              int m, int n, int k,
              const hipDoubleComplex *alpha,
              const hipDoubleComplex **A, int lda,
              const hipDoubleComplex **B, int ldb,
              const hipDoubleComplex *beta,
              hipDoubleComplex **C, int ldc,
              int batch_count)
{
    return hipblasZgemmBatched(*handle,transa,transb,m,n,k,alpha,A,lda,B,ldb,beta,C,ldc,batch_count);
}

extern "C" void f_cublasDestroy(hipblasHandle_t *handle)
{
    hipblasDestroy(*handle);
    free(handle);
}

extern "C" int f_cudaStreamCreate(hipStream_t **stream)
{
    *stream = (hipStream_t *) malloc(sizeof(hipStream_t));
    return hipStreamCreate(*stream);
}

extern "C" int f_cublasSetStream(hipblasHandle_t *handle, hipStream_t *streamid)
{
    return hipblasSetStream(*handle, *streamid);
}

extern "C" void f_cudaStreamDestroy(hipStream_t *stream)
{
    hipStreamDestroy(*stream);
}
